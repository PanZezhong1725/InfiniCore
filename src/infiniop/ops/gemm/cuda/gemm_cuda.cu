#include "../../../devices/cuda/cuda_handle.cuh"
#include "gemm_cuda.cuh"

namespace op::gemm::cuda {

struct Descriptor::Opaque {
    std::shared_ptr<device::cuda::Handle::Internal> internal;
};

Descriptor::~Descriptor() {
    delete _opaque;
}

infiniStatus_t Descriptor::create(
    infiniopHandle_t handle_,
    Descriptor **desc_ptr,
    infiniopTensorDescriptor_t c_desc,
    infiniopTensorDescriptor_t a_desc,
    infiniopTensorDescriptor_t b_desc) {
    auto handle = reinterpret_cast<device::cuda::nvidia::Handle *>(handle_);
    auto dtype = c_desc->dtype();

    if (dtype != INFINI_DTYPE_F16 && dtype != INFINI_DTYPE_F32) {
        return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }

    auto result = MatmulInfo::create(c_desc, a_desc, b_desc, MatrixLayout::COL_MAJOR);
    CHECK_RESULT(result);

    *desc_ptr = new Descriptor(
        dtype, result.take(), 0,
        new Opaque{handle->internal()},
        handle->device, handle->device_id);
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t Descriptor::calculate(
    void *workspace,
    size_t workspace_size,
    void *c,
    float beta,
    const void *a,
    const void *b,
    float alpha,
    void *stream) const {

    hipDataType a_type, b_type, c_type;
    hipblasComputeType_t compute_type;

    switch (_dtype) {
    case INFINI_DTYPE_F16:
        a_type = b_type = c_type = HIP_R_16F;
        compute_type = HIPBLAS_COMPUTE_32F;
        break;

    case INFINI_DTYPE_F32:
        a_type = b_type = c_type = HIP_R_32F;
#ifdef ENABLE_SUGON_CUDA_API
        compute_type = HIPBLAS_COMPUTE_32F;
#else
        compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
#endif
        break;

    default:
        return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }

    if (_info.is_transed) {
        std::swap(a, b);
    }

    auto op_a = _info.a_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    auto op_b = _info.b_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    CHECK_STATUS(_opaque->internal->useCublas(
        (hipStream_t)stream,
        [&](hipblasHandle_t handle) {
            CHECK_CUBLAS(
                hipblasGemmStridedBatchedEx(
                    handle,
                    op_a,
                    op_b,
                    static_cast<int>(_info.m),
                    static_cast<int>(_info.n),
                    static_cast<int>(_info.k),
                    &alpha,
                    a,
                    a_type,
                    static_cast<int>(_info.a_matrix.ld()),
                    _info.a_matrix.stride,
                    b,
                    b_type,
                    static_cast<int>(_info.b_matrix.ld()),
                    _info.b_matrix.stride,
                    &beta,
                    c,
                    c_type,
                    static_cast<int>(_info.c_matrix.ld()),
                    _info.c_matrix.stride,
                    static_cast<int>(_info.batch),
                    compute_type,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));
            return INFINI_STATUS_SUCCESS;
        }));
    return INFINI_STATUS_SUCCESS;
}

} // namespace op::gemm::cuda
