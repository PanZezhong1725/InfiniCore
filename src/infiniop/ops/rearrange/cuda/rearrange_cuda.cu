#include "hip/hip_runtime.h"
#include "../../../devices/cuda/cuda_common.cuh"
#include "../../../tensor.h"
#include "rearrange_cuda.cuh"
#include "rearrange_kernel.cuh"
#include <algorithm>
#include <cmath>
#include <memory>
#include <hip/hiprtc.h>
#include <stdint.h>
#include <vector>

namespace op::rearrange::cuda {

#define MAX_BLOCK_ARRAY_SIZE 5

#define MAX_GRID_ARRAY_SIZE 5

#define ARRAY_TYPE int

// 定义CUDA_CHECK宏
#define CUDA_CHECK(API)                                                   \
    do {                                                                  \
        hipError_t err = (API);                                          \
        if (err != hipSuccess) {                                         \
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err)); \
            std::abort();                                                 \
        }                                                                 \
    } while (0)

struct Descriptor::Opaque {
    std::shared_ptr<device::cuda::Handle::Internal> internal;
};

Descriptor::~Descriptor() {
    delete _opaque;
}

infiniStatus_t Descriptor::create(
    infiniopHandle_t handle,
    Descriptor **desc_ptr,
    infiniopTensorDescriptor_t y_desc,
    infiniopTensorDescriptor_t x_desc) {

    auto dtype = y_desc->dtype();
    auto ndim = y_desc->ndim();

    CHECK_API_OR(x_desc->dtype(), dtype, return INFINI_STATUS_BAD_TENSOR_DTYPE);
    CHECK_API_OR(x_desc->ndim(), ndim, return INFINI_STATUS_BAD_TENSOR_SHAPE);

    for (size_t i = 0; i < ndim; ++i) {
        CHECK_API_OR(x_desc->shape()[i], y_desc->shape()[i], return INFINI_STATUS_BAD_TENSOR_SHAPE);
    }

    // 保存临时vector对象
    auto y_shape = y_desc->shape();
    auto y_strides = y_desc->strides();
    auto x_strides = x_desc->strides();

    auto meta = utils::RearrangeMeta::create(
        y_shape.data(),
        y_strides.data(),
        x_strides.data(),
        ndim,
        infiniSizeOf(dtype));

    if (!meta) {
        return INFINI_STATUS_BAD_TENSOR_STRIDES;
    }

    *desc_ptr = new Descriptor(
        std::move(*meta),
        new Opaque{reinterpret_cast<device::cuda::Handle *>(handle)->internal()},
        handle->device, handle->device_id);
    return INFINI_STATUS_SUCCESS;
}

// 维度信息结构
struct Dim {
    size_t len;
    int src_stride;
    int dst_stride;
};

// 分割维度结构
struct SplitDim {
    size_t choose_idx;
    size_t num_per_block;
    size_t num_per_grid;
    int array_struct_idx_block;
    int array_struct_idx_grid;
    size_t dim_len;
};

// 根据元数据准备计算参数
RearrangeParams prepareRearrangeParams(const utils::RearrangeMeta &original_meta) {
    RearrangeParams params;

    // 获取更适合GPU处理的单元大小，这里使用2的幂次方
    auto meta_opt = original_meta.distribute_unit({32, 16, 8, 4, 2, 1});

    // 如果找不到合适的单元大小，直接panic
    if (!meta_opt.has_value()) {
        throw std::runtime_error("无法找到合适的单元大小");
    }
    const utils::RearrangeMeta &meta = meta_opt.value();

    // 获取维度信息
    const size_t ndim = meta.ndim();
    const size_t unit = meta.unit();

    // 特殊情况：无维度，只需要简单复制
    if (ndim == 0) {
        params.block_dim = 0;
        params.block_len_total = 1;
        params.block_len = {1};
        params.src_block_stride = {0};
        params.dst_block_stride = {0};
        params.grid_len = {1};
        params.src_grid_stride = {0};
        params.dst_grid_stride = {0};
        params.unit_size = unit;
        return params;
    }

    // 从元数据中提取必要的信息
    const ptrdiff_t *idx_strides = meta.idx_strides();
    const ptrdiff_t *dst_strides = meta.dst_strides();
    const ptrdiff_t *src_strides = meta.src_strides();

    // 准备维度信息
    std::vector<Dim> dims;
    std::vector<size_t> shape;
    dims.reserve(ndim);
    shape.reserve(ndim);

    auto prev_idx_stride = meta.count();
    for (size_t i = 0; i < ndim; ++i) {
        size_t len = prev_idx_stride / idx_strides[i];
        shape.push_back(len);
        dims.push_back({len, static_cast<int>(src_strides[i]), static_cast<int>(dst_strides[i])});
        prev_idx_stride = idx_strides[i];
    }

    // 计算src_strides的降序排序索引，类似于Rust版本中的src_strides_desc_idx
    std::vector<size_t> src_strides_desc_idx(ndim);
    for (size_t i = 0; i < ndim; ++i) {
        src_strides_desc_idx[i] = i;
    }
    std::sort(src_strides_desc_idx.begin(), src_strides_desc_idx.end(),
              [&dims](size_t a, size_t b) {
                  return std::abs(dims[a].src_stride) > std::abs(dims[b].src_stride);
              });

    // 根据最大线程数选择block和grid维度
    const size_t block_size = 256; // 与Rust版本保持一致
    std::vector<bool> block_dim_choose(ndim, false);

    // 初始化计数器
    size_t block_elements = 1;
    size_t block_src_elements = 1;
    size_t block_dst_elements = 1;
    size_t src_choose_idx = ndim;
    size_t dst_choose_idx = ndim;

    // 用于存储分割维度信息
    std::vector<SplitDim> split_dims;

    // 维度选择循环
    while (src_choose_idx > 0 && dst_choose_idx > 0) {
        // 获取当前需要处理的维度索引
        size_t src_idx = src_strides_desc_idx[src_choose_idx - 1];
        size_t dst_idx = dst_choose_idx - 1;

        if (src_idx == dst_idx) {
            // 源和目标维度相同，可以一起处理
            size_t idx = src_idx;
            size_t len = shape[idx];

            // 检查是否可以将此维度完全添加到block中
            if (block_elements * len <= block_size) {
                // 选择此维度
                block_dim_choose[idx] = true;
                block_elements *= len;
                block_src_elements *= len;
                block_dst_elements *= len;
                src_choose_idx--;
                dst_choose_idx--;
            } else {
                // 需要分割此维度
                size_t num_per_block = block_size / block_elements;

                // 确保num_per_block > 0且len >= num_per_block
                if (num_per_block > 0 && len >= num_per_block && num_per_block > 1) {
                    size_t num_per_grid = (len + num_per_block - 1) / num_per_block; // 向上取整

                    SplitDim split_dim = {
                        idx,           // choose_idx
                        num_per_block, // num_per_block
                        num_per_grid,  // num_per_grid
                        0,             // array_struct_idx_block (待更新)
                        0,             // array_struct_idx_grid (待更新)
                        len            // 原始维度长度
                    };
                    split_dims.push_back(split_dim);
                }
                break;
            }
        } else {
            // 源和目标维度不同，需要分别处理
            // 计算块比例
            double src_div_dst = static_cast<double>(block_src_elements) / block_dst_elements;
            double src_num_per_block = std::sqrt(block_size / (double)block_elements / src_div_dst);
            double dst_num_per_block = src_num_per_block * src_div_dst;

            size_t src_current_dim_len = shape[src_idx];
            size_t dst_current_dim_len = shape[dst_idx];

            if (static_cast<double>(src_current_dim_len) < src_num_per_block) {
                // 源维度可以完全添加到block
                block_dim_choose[src_idx] = true;
                block_elements *= src_current_dim_len;
                block_src_elements *= src_current_dim_len;
                src_choose_idx--;
            } else if (static_cast<double>(dst_current_dim_len) < dst_num_per_block) {
                // 目标维度可以完全添加到block
                block_dim_choose[dst_idx] = true;
                block_elements *= dst_current_dim_len;
                block_dst_elements *= dst_current_dim_len;
                dst_choose_idx--;
            } else {
                // 需要分割源和目标维度
                size_t src_num_per_block_int = static_cast<size_t>(std::floor(src_num_per_block));
                size_t dst_num_per_block_int = static_cast<size_t>(std::floor(dst_num_per_block));

                // 计算网格尺寸
                size_t src_num_per_grid = (src_current_dim_len + src_num_per_block_int - 1) / src_num_per_block_int; // 向上取整
                size_t dst_num_per_grid = (dst_current_dim_len + dst_num_per_block_int - 1) / dst_num_per_block_int; // 向上取整

                // 处理源维度
                if (src_num_per_block_int > 1) {
                    if (src_num_per_grid == 1) {
                        // 可以完全放入块
                        block_dim_choose[src_idx] = true;
                        block_elements *= src_current_dim_len;
                        block_src_elements *= src_current_dim_len;
                        src_choose_idx--;
                    } else {
                        // 需要分割
                        SplitDim split_dim = {
                            src_idx,               // choose_idx
                            src_num_per_block_int, // num_per_block
                            src_num_per_grid,      // num_per_grid
                            0,                     // array_struct_idx_block (待更新)
                            0,                     // array_struct_idx_grid (待更新)
                            src_current_dim_len    // 原始维度长度
                        };
                        split_dims.push_back(split_dim);
                    }
                }

                // 处理目标维度
                if (dst_num_per_block_int > 1) {
                    if (dst_num_per_grid == 1) {
                        // 可以完全放入块
                        block_dim_choose[dst_idx] = true;
                        block_elements *= dst_current_dim_len;
                        block_dst_elements *= dst_current_dim_len;
                        dst_choose_idx--;
                    } else {
                        // 需要分割
                        SplitDim split_dim = {
                            dst_idx,               // choose_idx
                            dst_num_per_block_int, // num_per_block
                            dst_num_per_grid,      // num_per_grid
                            0,                     // array_struct_idx_block (待更新)
                            0,                     // array_struct_idx_grid (待更新)
                            dst_current_dim_len    // 原始维度长度
                        };
                        split_dims.push_back(split_dim);
                    }
                }

                break;
            }
        }
    }

    // 准备block维度相关参数
    size_t block_dim = 0;
    size_t block_len_total = 1;

    std::vector<int> block_len;
    std::vector<int> src_block_stride;
    std::vector<int> dst_block_stride;

    std::vector<int> grid_len;
    std::vector<int> src_grid_stride;
    std::vector<int> dst_grid_stride;

    // 处理block维度，填充block_len和block_stride
    for (size_t i = 0; i < ndim; ++i) {
        if (block_dim_choose[i]) {
            block_len.push_back(shape[i]);
            src_block_stride.push_back(dims[i].src_stride);
            dst_block_stride.push_back(dims[i].dst_stride);
            block_dim += 1;
            block_len_total *= shape[i];
        }

        // 处理分割维度的block部分
        for (size_t j = 0; j < split_dims.size(); ++j) {
            if (i == split_dims[j].choose_idx) {
                block_len.push_back(split_dims[j].num_per_block);
                src_block_stride.push_back(dims[i].src_stride);
                dst_block_stride.push_back(dims[i].dst_stride);
                split_dims[j].array_struct_idx_block = block_dim;
                block_dim += 1;
                block_len_total *= split_dims[j].num_per_block;
            }
        }
    }

    // 处理grid维度，填充grid_len和grid_stride
    for (size_t i = 0; i < ndim; ++i) {
        if (!block_dim_choose[i]) {
            bool is_split = false;

            // 检查是否是分割维度
            for (size_t j = 0; j < split_dims.size(); ++j) {
                if (i == split_dims[j].choose_idx) {
                    is_split = true;
                    grid_len.push_back(split_dims[j].num_per_grid);
                    src_grid_stride.push_back(dims[i].src_stride * split_dims[j].num_per_block);
                    dst_grid_stride.push_back(dims[i].dst_stride * split_dims[j].num_per_block);
                    split_dims[j].array_struct_idx_grid = grid_len.size() - 1;
                }
            }

            // 如果不是分割维度，则作为完整的grid维度
            if (!is_split) {
                grid_len.push_back(shape[i]);
                src_grid_stride.push_back(dims[i].src_stride);
                dst_grid_stride.push_back(dims[i].dst_stride);
            }
        }
    }

    // 如果grid_len为空，添加一个默认值
    if (grid_len.empty()) {
        grid_len.push_back(1);
        src_grid_stride.push_back(0);
        dst_grid_stride.push_back(0);
    }

    // 处理约束条件 - 使用与Rust版本相似的逻辑
    std::vector<Constrains<int>> constrains;

    // 限制最多处理2个约束条件
    for (size_t i = 0; i < split_dims.size(); ++i) {
        if (split_dims[i].dim_len % split_dims[i].num_per_block == 0) {
            continue;
        }
        Constrains<int> constrain;
        constrain.grid_idx = split_dims[i].array_struct_idx_grid;
        constrain.block_idx = split_dims[i].array_struct_idx_block;
        constrain.grid_div_block = split_dims[i].num_per_block;
        constrain.total_len = split_dims[i].dim_len;
        constrains.push_back(constrain);
    }

    // 设置参数
    params.block_dim = block_dim;
    params.block_len_total = block_len_total;
    params.block_len = block_len;
    params.src_block_stride = src_block_stride;
    params.dst_block_stride = dst_block_stride;
    params.grid_len = grid_len;
    params.src_grid_stride = src_grid_stride;
    params.dst_grid_stride = dst_grid_stride;
    params.constrains = constrains;
    params.unit_size = unit;

    return params;
}

// 带约束的内核启动模板函数
template <unsigned int BLOCK_SIZE>
infiniStatus_t launchKernel(
    void *y,
    const void *x,
    unsigned int grid_size,
    const RearrangeParams &params,
    size_t unit_size,
    hipStream_t stream) {

    // 获取内核函数
    RearrangeParams params_copy = params; // 创建一个非const副本
    void *kernel_func = get_rearrange_kernel(params_copy);

    if (kernel_func == nullptr) {
        return INFINI_STATUS_BAD_PARAM;
    }

    // 创建非const的临时变量
    unsigned int block_dim = params.block_dim;
    unsigned int block_len_total = params.block_len_total;

    // 检查向量尺寸是否合理
    if (params.block_len.size() < block_dim || params.src_block_stride.size() < block_dim || params.dst_block_stride.size() < block_dim) {
        return INFINI_STATUS_BAD_PARAM;
    }

    if (params.grid_len.empty() || params.src_grid_stride.empty() || params.dst_grid_stride.empty()) {
        return INFINI_STATUS_BAD_PARAM;
    }

    const Constrains<int> *constrains_data;
    auto empty_constrains = Constrains<int>();
    if (params.constrains.empty()) {
        constrains_data = &empty_constrains;
    } else {
        constrains_data = params.constrains.data();
    }

    void *args[]
        = {
            &y, &x,
            &block_dim,
            &block_len_total,
            const_cast<void *>(static_cast<const void *>(params.block_len.data())),
            const_cast<void *>(static_cast<const void *>(params.src_block_stride.data())),
            const_cast<void *>(static_cast<const void *>(params.dst_block_stride.data())),
            const_cast<void *>(static_cast<const void *>(params.grid_len.data())),
            const_cast<void *>(static_cast<const void *>(params.src_grid_stride.data())),
            const_cast<void *>(static_cast<const void *>(params.dst_grid_stride.data())),
            const_cast<void *>(static_cast<const void *>(constrains_data))};

    try {
        CUDA_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(
            kernel_func),
            grid_size, BLOCK_SIZE,
            args, 0, stream));
    } catch (const std::exception &e) {
        return INFINI_STATUS_INTERNAL_ERROR;
    }

    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t Descriptor::calculate(
    void *y,
    const void *x,
    void *stream) const {

    auto hip_stream = reinterpret_cast<hipStream_t>(stream);

    // 如果没有维度，直接进行内存拷贝
    if (_meta.ndim() == 0) {
        CUDA_CHECK(hipMemcpyAsync(y, x, _meta.unit(), hipMemcpyDeviceToDevice, hip_stream));
        return INFINI_STATUS_SUCCESS;
    }

    // 准备参数
    RearrangeParams params = prepareRearrangeParams(_meta);

    // 计算grid大小
    unsigned int grid_size = 1;
    for (size_t i = 0; i < params.grid_len.size(); ++i) {
        grid_size *= params.grid_len[i];
    }

    // 检查grid大小是否为0
    if (grid_size == 0) {
        return INFINI_STATUS_BAD_PARAM;
    }

    // 获取设备属性
    int max_threads = _opaque->internal->maxThreadsPerBlock();

    // 根据设备属性选择合适的内核
    infiniStatus_t status = INFINI_STATUS_DEVICE_ARCHITECTURE_NOT_SUPPORTED;

    if (max_threads >= 1024) {
        status = launchKernel<1024>(y, x, grid_size, params, _meta.unit(), hip_stream);
    } else if (max_threads >= 512) {
        status = launchKernel<512>(y, x, grid_size, params, _meta.unit(), hip_stream);
    } else if (max_threads >= 256) {
        status = launchKernel<256>(y, x, grid_size, params, _meta.unit(), hip_stream);
    } else {
        return INFINI_STATUS_DEVICE_ARCHITECTURE_NOT_SUPPORTED;
    }

    // 检查内核启动状态
    if (status != INFINI_STATUS_SUCCESS) {

    } else {

        // 检查CUDA错误
        hipError_t cuda_err = hipGetLastError();
        if (cuda_err != hipSuccess) {
            return INFINI_STATUS_INTERNAL_ERROR;
        }
    }

    return status;
}

} // namespace op::rearrange::cuda
